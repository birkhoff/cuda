/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}



__global__ void add(int *a, int *b, int *c)
{

	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */

#define N 1024

int main(void)
{


	int *a, *b, *c; 				// host copies of a, b, c
	int *d_a, *d_b, *d_c; 			// device copies of a, b, c
	int size = N * sizeof(int);


	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Setup input values

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	int i = 0;
	for(i = 0; i < N; i++)
	{
		a[i] = i;//rand() % 100;
		b[i] = i;//rand() % 100;
	}

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	add<<<N,1>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(i = 0; i < N; i++)
	{
		printf("%d\n",c[i]);
	}


	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}